
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
#include <vector>
#include <functional>
#include <cstdlib>
#include <algorithm>

__global__ void matrixMul(const int *a, const int *b, int *c, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    c[row * N + col] = 0;
    for (int k = 0; k < N; k++)
    {
        c[row * N + col] += a[row * N + k] * b[k * N + col];
    }
}

int main()
{
    int N = 1 << 10;
    size_t bytes = N * N * sizeof(int);

    std::vector<int> h_a(N * N);
    std::vector<int> h_b(N * N);
    std::vector<int> h_c(N * N);

    std::generate(h_a.begin(), h_a.end(), []()
                  { return rand() % 100; });
    std::generate(h_b.begin(), h_b.end(), []()
                  { return rand() % 100; });

    int *d_a, *d_b, *d_c;

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

    int threads_num = 32;
    int blocks_num = N / threads_num;

    dim3 threads(threads_num, threads_num);
    dim3 blocks(blocks_num, blocks_num);

    matrixMul<<<blocks, threads>>>(d_a, d_b, d_c, N);

    hipMemcpy(h_c.data(),d_c,bytes,hipMemcpyDeviceToHost);
    
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            int tmp = 0;
            for (int k = 0; k < N; k++)
            {
                tmp += h_a[i * N + k] * h_b[k * N + j];
            }
            assert(tmp == h_c[i * N + j]);
        }
    }

    printf("run completed\n");
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}