
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cassert>
#include <vector>
#include <functional>
#include <algorithm>

__global__ void matrixMul(const int *__restrict a, const int *__restrict b, int *__restrict c, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    c[row * N + col] = 0;
    for (int k = 0; k < N; k++)
    {
        c[row * N + col] += a[row * N + k] * b[k * N + col];
    }
}

int main()
{
    int N = 1 << 10;
    size_t bytes = N * N * sizeof(int);

    std::vector<int> h_a(N * N);
    std::vector<int> h_b(N * N);
    std::vector<int> h_c(N * N);

    std::generate(h_a.begin(), h_a.end(), []()
                  { return rand() % 100; });

    std::generate(h_b.begin(), h_b.end(), []()
                  { return rand() % 100; });

    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

    int threads_num = 32;
    int blocks_num = N / threads_num;

    dim3 threads(threads_num,threads_num);
    dim3 blocks(blocks_num,blocks_num);

    
}